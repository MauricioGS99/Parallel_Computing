#include "hip/hip_runtime.h"
#include<stdio.h>
#define N 10

__global__ void add(int *a, int *b, int*c){
	int tid=blockIdx.x;
	if(tid<N)
			c[tid] = a[tid] + b[tid];
}

int main(void){
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;
	int i;

	//Allocate memory in GPU
	culdaMalloc((void***) &dev_a, N*sizeof(int));
	culdaMalloc((void***) &dev_b, N*sizeof(int));
	culdaMalloc((void***) &dev_c, N*sizeof(int));

	//Fill a and b
	for(i=0; i<N; i++){
	a[i]=i;
	b[i]=i*i;
	}

	//copy the arrays to the GPY
	hipMemcpy(dev_a,a,N*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_b,b,N*sizeof(int),hipMemcpyHostToDevice);

	//call to add:
	add<<N,1>>(dev_a,dev_b,dev_c); // N blocks of threads, 1 per block

	//copy the result array 'c' back from the GPU to the CPU
	hipMemcpy(c,dev_c,N*sizeof(int), cucaMemcpyDevicetoHost);

	//print c
		for(i=0;i<N,i++){
		printf("%d + %d = %d\n",a[i],b[i],c[i]);
	}
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;

}